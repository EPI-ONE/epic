#include "hip/hip_runtime.h"
// Cuckaroo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018-2019 Jiri Vadura (photon) and John Tromp
// This software is covered by the FAIR MINING license

#include "mean.cuh"

int gpuAssert(hipError_t code, char* file, int line, bool abort) {
    int device_id;
    hipGetDevice(&device_id);
    if (code != hipSuccess) {
        hipDeviceReset();
        if (abort) {
            spdlog::error("Device {} GPUassert({}): {} {} {}", device_id, code, hipGetErrorString(code), file, line);
            exit(code);
        }
    }
    return code;
}

#ifndef MAXSOLS
#define MAXSOLS 4
#endif

#ifndef XBITS
#define XBITS 6
#endif

const uint32_t NX     = 1 << XBITS;
const uint32_t NX2    = NX * NX;
const uint32_t XMASK  = NX - 1;
const uint32_t YBITS  = XBITS;
const uint32_t NY     = 1 << YBITS;
const uint32_t YZBITS = EDGEBITS - XBITS;
const uint32_t ZBITS  = YZBITS - YBITS;
const uint32_t NZ     = 1 << ZBITS;
const uint32_t ZMASK  = NZ - 1;

#ifndef IDXSHIFT
// number of bits of compression of surviving edge endpoints
// reduces space used in cycle finding, but too high a value
// results in NODE OVERFLOW warnings and fake cycles
#define IDXSHIFT 12
#endif

const uint32_t MAXEDGES = NEDGES >> IDXSHIFT;

#ifndef NEPS_A
#define NEPS_A 133
#endif
#ifndef NEPS_B
#define NEPS_B 88
#endif
#define NEPS 128

// Number of Parts of BufferB, all but one of which will overlap BufferA
#ifndef NB
#define NB 2
#endif

#ifndef NA
#define NA ((NB * NEPS_A + NEPS_B - 1) / NEPS_B)
#endif

const uint32_t EDGES_A = NZ * NEPS_A / NEPS;
const uint32_t EDGES_B = NZ * NEPS_B / NEPS;

const uint32_t ROW_EDGES_A = EDGES_A * NY;
const uint32_t ROW_EDGES_B = EDGES_B * NY;

__constant__ uint2 recoveredges[MAXCYCLELEN];
__constant__ uint2 e0 = {0, 0};

__device__ uint64_t dipblock(const siphash_keys& keys, const word_t edge, uint64_t* buf) {
    diphash_state<> shs(keys);
    word_t edge0 = edge & ~EDGE_BLOCK_MASK;
    uint32_t i;
    for (i = 0; i < EDGE_BLOCK_MASK; i++) {
        shs.hash24(edge0 + i);
        buf[i] = shs.xor_lanes();
    }
    shs.hash24(edge0 + i);
    buf[i] = 0;
    return shs.xor_lanes();
}

__device__ uint32_t endpoint(uint2 nodes, int uorv) {
    return uorv ? nodes.y : nodes.x;
}

#ifndef FLUSHA // should perhaps be in trimparams and passed as template
               // parameter
#define FLUSHA 16
#endif

template <int maxOut>
__global__ void SeedA(const siphash_keys& sipkeys, ulonglong4* __restrict__ buffer, uint32_t* __restrict__ indexes) {
    const int group    = blockIdx.x;
    const int dim      = blockDim.x;
    const int lid      = threadIdx.x;
    const int gid      = group * dim + lid;
    const int nthreads = gridDim.x * dim;
    const int FLUSHA2  = 2 * FLUSHA;

    __shared__ uint2 tmp[NX][FLUSHA2]; // needs to be ulonglong4 aligned
    const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(uint2);
    __shared__ int counters[NX];
    uint64_t buf[EDGE_BLOCK_SIZE];

    for (int row = lid; row < NX; row += dim)
        counters[row] = 0;
    __syncthreads();

    const int col   = group % NX;
    const int loops = NEDGES / nthreads; // assuming THREADS_HAVE_EDGES checked
    for (int blk = 0; blk < loops; blk += EDGE_BLOCK_SIZE) {
        uint32_t nonce0     = gid * loops + blk;
        const uint64_t last = dipblock(sipkeys, nonce0, buf);
        for (uint32_t e = 0; e < EDGE_BLOCK_SIZE; e++) {
            uint64_t edge     = buf[e] ^ last;
            uint32_t node0    = edge & EDGEMASK;
            uint32_t node1    = (edge >> 32) & EDGEMASK;
            int row           = node0 >> YZBITS;
            int counter       = min((int) atomicAdd(counters + row, 1),
                              (int) (FLUSHA2 - 1)); // assuming ROWS_LIMIT_LOSSES checked
            tmp[row][counter] = make_uint2(node0, node1);
            __syncthreads();
            if (counter == FLUSHA - 1) {
                int localIdx = min(FLUSHA2, counters[row]);
                int newCount = localIdx % FLUSHA;
                int nflush   = localIdx - newCount;
                uint32_t grp = row * NX + col;
                int cnt      = min((int) atomicAdd(indexes + grp, nflush), (int) (maxOut - nflush));
                for (int i = 0; i < nflush; i += TMPPERLL4)
                    buffer[((uint64_t) grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4*) (&tmp[row][i]);
                for (int t = 0; t < newCount; t++) {
                    tmp[row][t] = tmp[row][t + nflush];
                }
                counters[row] = newCount;
            }
            __syncthreads();
        }
    }
    uint2 zero = make_uint2(0, 0);
    for (int row = lid; row < NX; row += dim) {
        int localIdx = min(FLUSHA2, counters[row]);
        uint32_t grp = row * NX + col;
        for (int j = localIdx; j % TMPPERLL4; j++)
            tmp[row][j] = zero;
        for (int i = 0; i < localIdx; i += TMPPERLL4) {
            int cnt = min((int) atomicAdd(indexes + grp, TMPPERLL4), (int) (maxOut - TMPPERLL4));
            buffer[((uint64_t) grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4*) (&tmp[row][i]);
        }
    }
}

template <typename Edge>
__device__ bool null(Edge e);

__device__ bool null(uint32_t nonce) {
    return nonce == 0;
}

__device__ bool null(uint2 nodes) {
    return nodes.x == 0 && nodes.y == 0;
}

#ifndef FLUSHB
#define FLUSHB 8
#endif

template <typename T>
__device__ __forceinline__ T ldg(const T* ptr) {
#if __CUDA_ARCH__ >= 350
    return __ldg(ptr);
#else
    return *ptr;
#endif
}

template <int maxOut>
__global__ void SeedB(const uint2* __restrict__ source,
                      ulonglong4* __restrict__ destination,
                      const uint32_t* __restrict__ srcIdx,
                      uint32_t* __restrict__ dstIdx) {
    const int group   = blockIdx.x;
    const int dim     = blockDim.x;
    const int lid     = threadIdx.x;
    const int FLUSHB2 = 2 * FLUSHB;

    __shared__ uint2 tmp[NX][FLUSHB2];
    const int TMPPERLL4 = sizeof(ulonglong4) / sizeof(uint2);
    __shared__ int counters[NX];

    for (int col = lid; col < NX; col += dim)
        counters[col] = 0;
    __syncthreads();
    const int row         = group / NX;
    const int bucketEdges = min((int) srcIdx[group], (int) maxOut);
    const int loops       = (bucketEdges + dim - 1) / dim;
    for (int loop = 0; loop < loops; loop++) {
        int col;
        int counter         = 0;
        const int edgeIndex = loop * dim + lid;
        if (edgeIndex < bucketEdges) {
            const int index = group * maxOut + edgeIndex;
            uint2 edge      = ldg(&source[index]);
            if (!null(edge)) {
                uint32_t node1    = edge.x;
                col               = (node1 >> ZBITS) & XMASK;
                counter           = min((int) atomicAdd(counters + col, 1),
                              (int) (FLUSHB2 - 1)); // assuming COLS_LIMIT_LOSSES checked
                tmp[col][counter] = edge;
            }
        }
        __syncthreads();
        if (counter == FLUSHB - 1) {
            int localIdx = min(FLUSHB2, counters[col]);
            int newCount = localIdx % FLUSHB;
            int nflush   = localIdx - newCount;
            uint32_t grp = row * NX + col;
            int cnt      = min((int) atomicAdd(dstIdx + grp, nflush), (int) (maxOut - nflush));
            for (int i = 0; i < nflush; i += TMPPERLL4)
                destination[((uint64_t) grp * maxOut + cnt + i) / TMPPERLL4] = *(ulonglong4*) (&tmp[col][i]);
            for (int t = 0; t < newCount; t++) {
                tmp[col][t] = tmp[col][t + nflush];
            }
            counters[col] = newCount;
        }
        __syncthreads();
    }
    uint2 zero = make_uint2(0, 0);
    for (int col = lid; col < NX; col += dim) {
        int localIdx = min(FLUSHB2, counters[col]);
        uint32_t grp = row * NX + col;
        for (int j = localIdx; j % TMPPERLL4; j++)
            tmp[col][j] = zero;
        for (int i = 0; i < localIdx; i += TMPPERLL4) {
            int cnt = min((int) atomicAdd(dstIdx + grp, TMPPERLL4), (int) (maxOut - TMPPERLL4));
            destination[((uint64_t) grp * maxOut + cnt) / TMPPERLL4] = *(ulonglong4*) (&tmp[col][i]);
        }
    }
}

__device__ __forceinline__ void Increase2bCounter(uint32_t* ecounters, const int bucket) {
    int word          = bucket >> 5;
    unsigned char bit = bucket & 0x1F;
    uint32_t mask     = 1 << bit;

    uint32_t old = atomicOr(ecounters + word, mask) & mask;
    if (old)
        atomicOr(ecounters + word + NZ / 32, mask);
}

__device__ __forceinline__ bool Read2bCounter(uint32_t* ecounters, const int bucket) {
    int word          = bucket >> 5;
    unsigned char bit = bucket & 0x1F;

    return (ecounters[word + NZ / 32] >> bit) & 1;
}

template <int NP, int maxIn, int maxOut>
__global__ void Round(const int round,
                      const uint2* __restrict__ src,
                      uint2* __restrict__ dst,
                      const uint32_t* __restrict__ srcIdx,
                      uint32_t* __restrict__ dstIdx) {
    const int group        = blockIdx.x;
    const int dim          = blockDim.x;
    const int lid          = threadIdx.x;
    const int COUNTERWORDS = NZ / 16; // 16 2-bit counters per 32-bit word

    __shared__ uint32_t ecounters[COUNTERWORDS];

    for (int i = lid; i < COUNTERWORDS; i += dim)
        ecounters[i] = 0;
    __syncthreads();

    for (int i = 0; i < NP; i++, src += NX2 * maxIn, srcIdx += NX2) {
        const int edgesInBucket = min(srcIdx[group], maxIn);
        const int loops         = (edgesInBucket + dim - 1) / dim;

        for (int loop = 0; loop < loops; loop++) {
            const int lindex = loop * dim + lid;
            if (lindex < edgesInBucket) {
                const int index = maxIn * group + lindex;
                uint2 edge      = ldg(&src[index]);
                if (null(edge))
                    continue;
                uint32_t node = endpoint(edge, round & 1);
                Increase2bCounter(ecounters, node & ZMASK);
            }
        }
    }

    __syncthreads();

    src -= NP * NX2 * maxIn;
    srcIdx -= NP * NX2;
    for (int i = 0; i < NP; i++, src += NX2 * maxIn, srcIdx += NX2) {
        const int edgesInBucket = min(srcIdx[group], maxIn);
        const int loops         = (edgesInBucket + dim - 1) / dim;
        for (int loop = 0; loop < loops; loop++) {
            const int lindex = loop * dim + lid;
            if (lindex < edgesInBucket) {
                const int index = maxIn * group + lindex;
                uint2 edge      = ldg(&src[index]);
                if (null(edge))
                    continue;
                uint32_t node0 = endpoint(edge, round & 1);
                if (Read2bCounter(ecounters, node0 & ZMASK)) {
                    uint32_t node1                = endpoint(edge, (round & 1) ^ 1);
                    const int bucket              = node1 >> ZBITS;
                    const int bktIdx              = min(atomicAdd(dstIdx + bucket, 1), maxOut - 1);
                    dst[bucket * maxOut + bktIdx] = (round & 1) ? make_uint2(node1, node0) : make_uint2(node0, node1);
                }
            }
        }
    }
}

template <int maxIn>
__global__ void Tail(const uint2* source, uint2* destination, const uint32_t* srcIdx, uint32_t* dstIdx) {
    const int lid   = threadIdx.x;
    const int group = blockIdx.x;
    const int dim   = blockDim.x;
    int myEdges     = srcIdx[group];
    __shared__ int destIdx;

    if (lid == 0)
        destIdx = atomicAdd(dstIdx, myEdges);
    __syncthreads();
    for (int i = lid; i < myEdges; i += dim)
        destination[destIdx + lid] = source[group * maxIn + lid];
}

__global__ void Recovery(const siphash_keys& sipkeys, ulonglong4* buffer, int* indexes, int proofsize) {
    const int gid      = blockDim.x * blockIdx.x + threadIdx.x;
    const int lid      = threadIdx.x;
    const int nthreads = blockDim.x * gridDim.x;
    const int loops    = NEDGES / nthreads;
    __shared__ uint32_t nonces[MAXCYCLELEN];
    uint64_t buf[EDGE_BLOCK_SIZE];

    if (lid < proofsize) {
        nonces[lid] = 0;
    }
    __syncthreads();
    for (int blk = 0; blk < loops; blk += EDGE_BLOCK_SIZE) {
        uint32_t nonce0     = gid * loops + blk;
        const uint64_t last = dipblock(sipkeys, nonce0, buf);
        for (int i = 0; i < EDGE_BLOCK_SIZE; i++) {
            uint64_t edge = buf[i] ^ last;
            uint32_t u    = edge & EDGEMASK;
            uint32_t v    = (edge >> 32) & EDGEMASK;
            for (int p = 0; p < proofsize; p++) { // YO
                if (recoveredges[p].x == u && recoveredges[p].y == v) {
                    nonces[p] = nonce0 + i;
                }
            }
        }
    }
    __syncthreads();
    if (lid < proofsize) {
        if (nonces[lid] > 0) {
            indexes[lid] = nonces[lid];
        }
    }
}

trimparams::trimparams() {
    ntrims         = 176;
    genA.blocks    = 4096;
    genA.tpb       = 256;
    genB.blocks    = NX2;
    genB.tpb       = 128;
    trim.blocks    = NX2;
    trim.tpb       = 512;
    tail.blocks    = NX2;
    tail.tpb       = 1024;
    recover.blocks = 1024;
    recover.tpb    = 1024;
}

GEdgeTrimmer::GEdgeTrimmer(const trimparams _tp, int cyclelen)
    : cycle_len(cyclelen), tp(_tp), indexesSize(NX * NY * sizeof(uint32_t)) /*, indexesE(new uint32_t*[1 + NB])*/ {
    checkCudaErrors_V(hipMalloc((void**) &dt, sizeof(GEdgeTrimmer)));
    checkCudaErrors_V(hipMalloc((void**) &uvnodes, cycle_len * 2 * sizeof(uint32_t)));
    checkCudaErrors_V(hipMalloc((void**) &dipkeys, sizeof(siphash_keys)));
    for (int i = 0; i < 1 + NB; i++) {
        checkCudaErrors_V(hipMalloc((void**) &indexesE[i], indexesSize));
    }
    sizeA                   = ROW_EDGES_A * NX * sizeof(uint2);
    sizeB                   = ROW_EDGES_B * NX * sizeof(uint2);
    const size_t bufferSize = sizeA + sizeB / NB;
    assert(bufferSize >= sizeB + sizeB / NB / 2); // ensure enough space for Round 1
    checkCudaErrors_V(hipMalloc((void**) &bufferA, bufferSize));
    bufferAB = bufferA + sizeB / NB;
    bufferB  = bufferA + bufferSize - sizeB;
    assert(bufferA + sizeA == bufferB + sizeB * (NB - 1) / NB); // ensure alignment of overlap
    hipMemcpy(dt, this, sizeof(GEdgeTrimmer), hipMemcpyHostToDevice);
    initsuccess = true;
}

uint64_t GEdgeTrimmer::globalbytes() const {
    return (sizeA + sizeB / NB) + (1 + NB) * indexesSize + sizeof(siphash_keys) + cycle_len * 2 * sizeof(uint32_t) +
           sizeof(GEdgeTrimmer);
}

GEdgeTrimmer::~GEdgeTrimmer() {
    checkCudaErrors_V(hipFree(bufferA));
    for (int i = 0; i < 1 + NB; i++) {
        checkCudaErrors_V(hipFree(indexesE[i]));
    }
    /*delete[] indexesE;*/
    checkCudaErrors_V(hipFree(dipkeys));
    checkCudaErrors_V(hipFree(uvnodes));
    checkCudaErrors_V(hipFree(dt));
    hipDeviceReset();
}

uint32_t GEdgeTrimmer::trim() {
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    hipMemcpy(dipkeys, &sipkeys, sizeof(sipkeys), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    float durationA, durationB;
    hipEventRecord(start, NULL);

    hipMemset(indexesE[1], 0, indexesSize);

    SeedA<EDGES_A><<<tp.genA.blocks, tp.genA.tpb>>>(*dipkeys, (ulonglong4*) bufferAB, indexesE[1]);

    checkCudaErrors(hipDeviceSynchronize());
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&durationA, start, stop);
    if (abort.load()) {
        return false;
    }
    hipEventRecord(start, NULL);

    hipMemset(indexesE[0], 0, indexesSize);

    uint32_t qA = sizeA / NA;
    uint32_t qE = NX2 / NA;
    for (uint32_t i = 0; i < NA; i++) {
        SeedB<EDGES_A><<<tp.genB.blocks / NA, tp.genB.tpb>>>(
            (uint2*) (bufferAB + i * qA), (ulonglong4*) (bufferA + i * qA), indexesE[1] + i * qE, indexesE[0] + i * qE);
        if (abort.load()) {
            return false;
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&durationB, start, stop);
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    spdlog::trace("Seeding completed in {} + {} ms", durationA, durationB);
    if (abort.load()) {
        return false;
    }

    for (uint32_t i = 0; i < NB; i++)
        hipMemset(indexesE[1 + i], 0, indexesSize);

    qA              = sizeA / NB;
    const size_t qB = sizeB / NB;
    qE              = NX2 / NB;
    for (uint32_t i = NB; i--;) {
        Round<1, EDGES_A, EDGES_B / NB>
            <<<tp.trim.blocks / NB, tp.trim.tpb>>>(0, (uint2*) (bufferA + i * qA), (uint2*) (bufferB + i * qB),
                                                   indexesE[0] + i * qE, indexesE[1 + i]); // to .632
        if (abort.load()) {
            return false;
        }
    }

    hipMemset(indexesE[0], 0, indexesSize);

    Round<NB, EDGES_B / NB, EDGES_B / 2><<<tp.trim.blocks, tp.trim.tpb>>>(1, (const uint2*) bufferB, (uint2*) bufferA,
                                                                          indexesE[1],
                                                                          indexesE[0]); // to .296
    if (abort.load()) {
        return false;
    }

    hipMemset(indexesE[1], 0, indexesSize);

    Round<1, EDGES_B / 2, EDGES_A / 4><<<tp.trim.blocks, tp.trim.tpb>>>(2, (const uint2*) bufferA, (uint2*) bufferB,
                                                                        indexesE[0],
                                                                        indexesE[1]); // to .176
    if (abort.load()) {
        return false;
    }

    hipMemset(indexesE[0], 0, indexesSize);

    Round<1, EDGES_A / 4, EDGES_B / 4><<<tp.trim.blocks, tp.trim.tpb>>>(3, (const uint2*) bufferB, (uint2*) bufferA,
                                                                        indexesE[1],
                                                                        indexesE[0]); // to .117
    if (abort.load()) {
        return false;
    }

    hipDeviceSynchronize();

    for (int round = 4; round < tp.ntrims; round += 2) {
        hipMemset(indexesE[1], 0, indexesSize);
        Round<1, EDGES_B / 4, EDGES_B / 4><<<tp.trim.blocks, tp.trim.tpb>>>(round, (const uint2*) bufferA,
                                                                            (uint2*) bufferB, indexesE[0], indexesE[1]);
        if (abort.load()) {
            return false;
        }
        hipMemset(indexesE[0], 0, indexesSize);
        Round<1, EDGES_B / 4, EDGES_B / 4><<<tp.trim.blocks, tp.trim.tpb>>>(round + 1, (const uint2*) bufferB,
                                                                            (uint2*) bufferA, indexesE[1], indexesE[0]);
        if (abort.load()) {
            return false;
        }
    }

    hipMemset(indexesE[1], 0, indexesSize);
    hipDeviceSynchronize();

    Tail<EDGES_B / 4>
        <<<tp.tail.blocks, tp.tail.tpb>>>((const uint2*) bufferA, (uint2*) bufferB, indexesE[0], indexesE[1]);
    hipMemcpy(&nedges, indexesE[1], sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return nedges;
}

SolverCtx::SolverCtx(const trimparams& tp, int cyclelen)
    : trimmer(tp, 0), cg(MAXEDGES, MAXEDGES, MAXSOLS, IDXSHIFT, cyclelen) {
    cycle_len = cyclelen;
    edges     = new uint2[MAXEDGES];
    soledges  = new uint2[cycle_len];
}

int SolverCtx::findcycles(uint2* edges, uint32_t nedges) {
    cg.reset();
    for (uint32_t i = 0; i < nedges; i++) {
        cg.add_compress_edge(edges[i].x, edges[i].y);
    }

    for (uint32_t s = 0; s < cg.nsols; s++) {
        for (uint32_t j = 0; j < cycle_len; j++) {
            soledges[j] = edges[cg.sols[s][j]];
        }
        sols.resize(sols.size() + cycle_len);
        hipMemcpyToSymbol(HIP_SYMBOL(recoveredges), soledges, sizeof(uint2) * cycle_len);
        hipMemset(trimmer.indexesE[1], 0, trimmer.indexesSize);
        Recovery<<<trimmer.tp.recover.blocks, trimmer.tp.recover.tpb>>>(*trimmer.dipkeys, (ulonglong4*) trimmer.bufferA,
                                                                        (int*) trimmer.indexesE[1], cycle_len);
        hipMemcpy(&sols[sols.size() - cycle_len], trimmer.indexesE[1], cycle_len * sizeof(uint32_t),
                   hipMemcpyDeviceToHost);
        checkCudaErrors(hipDeviceSynchronize());
        qsort(&sols[sols.size() - cycle_len], cycle_len, sizeof(uint32_t), cg.nonce_cmp);
    }

    return 0;
}

int SolverCtx::solve() {
    trimmer.abort   = false;
    uint32_t nedges = trimmer.trim();
    if (!nedges)
        return 0;
    if (nedges > MAXEDGES) {
        spdlog::trace("OOPS; losing {} edges beyond MAXEDGES={}", nedges - MAXEDGES, MAXEDGES);
        nedges = MAXEDGES;
    }
    hipMemcpy(edges, trimmer.bufferB, sizeof(uint2[nedges]), hipMemcpyDeviceToHost);
    findcycles(edges, nedges);
    spdlog::trace("findcycles edges {}", nedges);
    return sols.size() / cycle_len;
}

void FillDefaultGPUParams(SolverParams& params) {
    trimparams tp;
    params.device        = 0;
    params.ntrims        = tp.ntrims;
    params.genablocks    = min(tp.genA.blocks, NEDGES / EDGE_BLOCK_SIZE / tp.genA.tpb);
    params.genatpb       = tp.genA.tpb;
    params.genbtpb       = tp.genB.tpb;
    params.trimtpb       = tp.trim.tpb;
    params.tailtpb       = tp.tail.tpb;
    params.recoverblocks = min(tp.recover.blocks, NEDGES / EDGE_BLOCK_SIZE / tp.recover.tpb);
    params.recovertpb    = tp.recover.tpb;
    params.cpuload       = false;
}

SolverCtx* CreateSolverCtx(SolverParams& params, int cyclelen) {
    trimparams tp;
    tp.ntrims         = params.ntrims;
    tp.genA.blocks    = params.genablocks;
    tp.genA.tpb       = params.genatpb;
    tp.genB.tpb       = params.genbtpb;
    tp.trim.tpb       = params.trimtpb;
    tp.tail.tpb       = params.tailtpb;
    tp.recover.blocks = params.recoverblocks;
    tp.recover.tpb    = params.recovertpb;

    hipDeviceProp_t prop;
    checkCudaErrors_N(hipGetDeviceProperties(&prop, params.device));

    assert(tp.genA.tpb <= prop.maxThreadsPerBlock);
    assert(tp.genB.tpb <= prop.maxThreadsPerBlock);
    assert(tp.trim.tpb <= prop.maxThreadsPerBlock);
    // assert(tp.tailblocks <= prop.threadDims[0]);
    assert(tp.tail.tpb <= prop.maxThreadsPerBlock);
    assert(tp.recover.tpb <= prop.maxThreadsPerBlock);

    assert(tp.genA.blocks * tp.genA.tpb * EDGE_BLOCK_SIZE <= NEDGES);       // check THREADS_HAVE_EDGES
    assert(tp.recover.blocks * tp.recover.tpb * EDGE_BLOCK_SIZE <= NEDGES); // check THREADS_HAVE_EDGES
    assert(tp.genA.tpb / NX <= FLUSHA);                                     // check ROWS_LIMIT_LOSSES
    assert(tp.genB.tpb / NX <= FLUSHB);                                     // check COLS_LIMIT_LOSSES

    checkCudaErrors_N(hipSetDevice(params.device));
    if (!params.cpuload) {
        checkCudaErrors_N(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
    }

    return new SolverCtx(tp, cyclelen);
}
